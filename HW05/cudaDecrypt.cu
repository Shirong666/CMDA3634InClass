#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//compute a*b mod p safely
__device__ unsigned int kermodprod(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int kermodExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = kermodprod(aExpb, z, p);
    z = kermodprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}





__global__ void gus(unsigned int *p, unsigned int *g, unsigned int *h, unsigned int *x)
{
int threadid = threadIdx.x;
int blockid = blockIdx.x;
int Nblock = blockDim.x;

int id = threadid+blockid*Nblock;
if (id < *p-1)
{
if (modExp(*g, id+1,*p) == *h)
{
    *x = id +1;
    printf("Secret key found! x = %u \n", id +1);
}
}
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */
//declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  FILE* public_key;
  public_key = fopen("public_key.txt",
             "r");

  FILE * mes;
  mes = fopen("message.txt","r");
  fscanf(public_key,"%u\n%u\n%u\n%u\n",
                &n,&p,&g,&h);

 fscanf(mes,"%u\n",&Nints);

  unsigned int *Zmessage =
             (unsigned int *) malloc(Nints*
                      sizeof(unsigned int));


unsigned int *a =
           (unsigned int *) malloc(Nints*
            sizeof(unsigned int));

for (unsigned int i=0;i<Nints;i++) {
         fscanf(mes,"%u %u\n",
                  &Zmessage[i], &a[i]);
            }

fclose(public_key);
fclose(mes);

unsigned int *d_p, *d_g, *d_h, *d_x;

hipMalloc(&d_p, sizeof(unsigned int));
hipMalloc(&d_g, sizeof(unsigned int));
hipMalloc(&d_h, sizeof(unsigned int));
hipMalloc(&d_x, sizeof(unsigned int));



hipMemcpy(d_p, &p, sizeof(unsigned int),hipMemcpyHostToDevice);
 hipMemcpy(d_g, &g, sizeof(unsigned int),
          hipMemcpyHostToDevice);
  hipMemcpy(d_h, &h, sizeof(unsigned int),
           hipMemcpyHostToDevice);
   

int Nthreads = 1024;
int Nblocks = (p-1+Nthreads-1)/Nthreads;

 // find the secret key
if (x==0 || modExp(g,x<Plug>PeepOpen)!=h) {
      printf("Finding the secret key...\n");
      double startTime = clock();

      gus <<<Nblocks, Nthreads >>>(d_p,d_g,d_h,d_x);

 
      hipMemcpy(&x, d_x, sizeof(unsigned int),hipMemcpyDeviceToHost);


 //   for (unsigned int i=0;i<p-1;i++) {
 //     if (modExp(g,i+1,p)==h) {
 //       printf("Secret key found! x = %u \n", i+1);
 //       x=i+1;
 //     } 
 //   }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);

    hipFree(d_p);
    hipFree(d_g);
    hipFree(d_h);
    hipFree(d_x);
    


}

  /* Q3 After finding the secret key, decrypt the message */

  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));

 ElGamalDecrypt(Zmessage,a,Nints,p,x);

 convertZToString(Zmessage, Nints, message, Nints*(n-1)/8);


  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
